#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024
#define BLOCK_SIZE 16
#define N_STREAMS 4

__global__ void matmul(int *a, int *b, int *c, int offset)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if ((row < N) && (col < N))
    {
        c[(row * N + col) + offset] = 0;
        for (int k = 0; k < N; k++)
        {
            c[(row * N + col) + offset] += a[(row * N + k) + offset] * b[k * N + col];
        }
    }
}

int main()
{
    int *h_a, *h_a2, *h_b, *h_c;
    int *d_a, *d_a2, *d_b, *d_c;
    int i, j;
    float ms;
    hipStream_t streams[N_STREAMS];
    hipEvent_t startEvent, stopEvent;

    for (i = 0; i < N_STREAMS; i++)
        hipStreamCreate(&streams[i]);

    hipEventCreateWithFlags(&startEvent, 0);
    hipEventCreateWithFlags(&stopEvent, 0);

    // Allocate host memory
    hipHostMalloc(&h_a, N * N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_a2, N * N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_b, N * N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_c, N * N * sizeof(int), hipHostMallocDefault);

    hipMalloc(&d_a, N * N * sizeof(int));
    hipMalloc(&d_a2, N * N * sizeof(int));
    hipMalloc(&d_b, N * N * sizeof(int));
    hipMalloc(&d_c, N * N * sizeof(int));

    // Initialize the host arrays
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            h_a[i * N + j] = i;
            h_a2[i * N + j] = i;
            h_b[i * N + j] = j;
        }
    }

    dim3 grid(N / (BLOCK_SIZE * N_STREAMS), N / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    hipEventRecord(startEvent, 0);

    // Synchronously copy b to device
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Async

    hipMemcpyAsync(&d_a[0 * (N * N / N_STREAMS)], &h_a[0 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(&d_a[1 * (N * N / N_STREAMS)], &h_a[1 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyHostToDevice, streams[1]);
    hipMemcpyAsync(&d_a[2 * (N * N / N_STREAMS)], &h_a[2 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyHostToDevice, streams[2]);
    hipMemcpyAsync(&d_a[3 * (N * N / N_STREAMS)], &h_a[3 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyHostToDevice, streams[3]);

    matmul<<<grid, block, 0, streams[0]>>>(d_a, d_b, d_c, 0 * (N * N / N_STREAMS));
    matmul<<<grid, block, 0, streams[1]>>>(d_a, d_b, d_c, 1 * (N * N / N_STREAMS));
    matmul<<<grid, block, 0, streams[2]>>>(d_a, d_b, d_c, 2 * (N * N / N_STREAMS));
    matmul<<<grid, block, 0, streams[3]>>>(d_a, d_b, d_c, 3 * (N * N / N_STREAMS));

    hipMemcpyAsync(&h_c[0 * (N * N / N_STREAMS)], &d_c[0 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyDeviceToHost, streams[3]);
    hipMemcpyAsync(&h_c[1 * (N * N / N_STREAMS)], &d_c[1 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyDeviceToHost, streams[2]);
    hipMemcpyAsync(&h_c[2 * (N * N / N_STREAMS)], &d_c[2 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyDeviceToHost, streams[1]);
    hipMemcpyAsync(&h_c[3 * (N * N / N_STREAMS)], &d_c[3 * (N * N / N_STREAMS)], N * (N / N_STREAMS) * sizeof(int), hipMemcpyDeviceToHost, streams[0]);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("N Streams: %d\n", N_STREAMS);
    printf("Time taken: %fms\n", ms);
    printf("C[451][451]: %d\n", h_c[451 * N + 451]);

    // Cleanup
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    for (i = 0; i < N_STREAMS; i++)
        hipStreamDestroy(streams[i]);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}